#include "hip/hip_runtime.h"
#include<iostream>
#include "hip/hip_runtime.h"
#include ""
using namespace std;

#define N 5
#define INF 99999

// Device code
__global__ void matrixFloyd(float* graph, int k)
{
    int i = blockIdx.x + threadIdx.x;
	int j = blockIdx.y + threadIdx.y;
	
    if(graph[procid][k] + graph[k][j] < graph[procid][j])
		graph[procid][j] = graph[procid][k] + graph[k][j];
}
            
// Host code
int main()
{
    size_t size = N * N * sizeof(int);

    // Allocate input vectors h_A and h_B in host memory

    // Initialize input vectors
	int h_graph[N][N] = {
		0,3,9,8,3,
		5,0,1,4,2,
		6,6,0,4,5,
		2,9,2,0,7,
		7,9,3,2,0,
	};
	
    // Allocate vectors in device memory
    int* d_graph;
    hipMalloc(&d_graph, size);
	
    // Copy vectors from host memory to device memory
    hipMemcpy(d_graph, h_graph, size, hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 threadsPerBlock(N, N);
	int numBlocks= 1;

	for(int k = 0; k < N; k++)
	{
		VecAdd<<<numBlocks, threadsPerBlock>>>(d_graph, k);

	}
		
    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_graph, d_graph, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_graph);
            
    // Free host memory
    for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			if (h_graph[i][j] == INF)
				cout << "INF ";
			else
				cout << h_graph[i][j] << " ";
		}
		cout << endl;
	}
}
